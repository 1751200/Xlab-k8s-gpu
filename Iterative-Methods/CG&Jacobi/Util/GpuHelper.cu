#include "GpuHelper.cuh"
 void GpuHelper::selectGpu(int *devsNum, int *gpuNum) {
	int best = *gpuNum;   // �õ�ϵͳ��NVIDIA GPU����Ŀ
	hipGetDeviceCount(devsNum);
	if (*devsNum > 1) {
		int devId;
		int maxCores = 0;
		for (devId = 0; devId < *devsNum; devId++) {
			hipDeviceProp_t devProperties;
			hipGetDeviceProperties(&devProperties, devId);
			if (maxCores < devProperties.multiProcessorCount) {
				maxCores = devProperties.multiProcessorCount;//���ദ����(SM)�ĸ���
				best = devId;
			}
		}
		*gpuNum = best;
	}
}

void GpuHelper::testDevice(int devId) {
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, devId);
	//CUDA Capability Major/Minor version number
	if (devProp.major == 9999 && devProp.minor == 9999) {
		//printf("No device supporting CUDA.\n");
		hipDeviceReset();
	}
	else
		//printf("Using GPU device number %d.\n", devId);
		return;
}