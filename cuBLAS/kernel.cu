#include "hip/hip_runtime.h"

// CUDA runtime �� + CUBLAS ��
#include <hipblas.h>
#include "hip/hip_runtime.h"
#include ""
#include"hip/device_functions.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include<iostream>
using namespace std;

using namespace std;

// ������Ծ����ά��
int  sizeT = 4096;
int  aW = sizeT;
int  aH = sizeT;
int  bW = sizeT;
int  bH = sizeT;
int  cW = bW;
int  cH = aH;
void mul_cpu(float * a, float * b, float * c) {
	for (int i = 0; i < aH; i++) {
		for (int j = 0; j < bW; j++) {
			c[i*bW + j] = 0;
			for (int k = 0; k < aW; k++) {
				c[i*bW + j] += (a[i*aW + k]) * (b[k*bW + j]);
			}
		}
	}
}
float check_diff(float * gpuC, float * cpuC) {
	float diff = 0;
	for (int i = 0; i < aH*bW; i++) {
		diff += abs(gpuC[i] - cpuC[i]);
	}
	return diff;
}

int matrix()
{
	// ����״̬����
	hipblasStatus_t status;

	// �� �ڴ� ��Ϊ��Ҫ����ľ��󿪱ٿռ�
	float *h_A = (float*)malloc(aW*aH * sizeof(float));
	float *h_B = (float*)malloc(bW*bH * sizeof(float));

	// �� �ڴ� ��Ϊ��Ҫ����������ľ��󿪱ٿռ�
	float *h_C = (float*)malloc(cW*cH * sizeof(float));

	// �� �ڴ� ��Ϊ��Ҫ����������ľ��󿪱ٿռ�
	float *cpu_C = (float*)malloc(cW*cH * sizeof(float));

	//Ϊ����������Ԫ�ظ�ֵ
	for (int i = 0; i < aW*aH; ++i)
	{
		h_A[i] = (float)(rand() / (float)RAND_MAX);//(float)(rand() % 10);// (float)(rand() / (float)RAND_MAX); //(float)(rand() %10); (float)i;
	}
	for (int i = 0; i < bW*bH; ++i)
	{
		h_B[i] = (float)(rand() / (float)RAND_MAX);//(float)(rand() % 10);//  //(float)i*i;
	}
	//h_A[0] = 1; h_A[1] = 1;
	//h_B[0] = 2; h_B[1] = 1; h_B[2] = 0; h_B[3] = 1; h_B[4] = 3; h_B[5] = 0;



	//// ��ӡ�����Եľ���
	//cout << "���� A :" << endl;
	//for (int i = 0; i < aW*aH; i++) {
	//	cout << h_A[i] << " ";
	//	if ((i + 1) % aW == 0) cout << endl;
	//}
	//cout << endl;
	//cout << "���� B :" << endl;
	//for (int i = 0; i < bW*bH; i++) {
	//	cout << h_B[i] << " ";
	//	if ((i + 1) % bW == 0) cout << endl;
	//}
	//cout << endl;

	/*
	** GPU ����������
	*/

	
	float elapsedTime = 0.0;
	hipEvent_t event_start, event_stop;
	hipEventCreate(&event_start);
	hipEventCreate(&event_stop);
	hipEventRecord(event_start, 0);

	// ��������ʼ�� CUBLAS �����
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);

	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
			cout << "CUBLAS ����ʵ��������" << endl;
		}
		getchar();
		return EXIT_FAILURE;
	}



	float *d_A, *d_B, *d_C;
	// �� �Դ� ��Ϊ��Ҫ����ľ��󿪱ٿռ�
	hipMalloc(
		(void**)&d_A,    // ָ�򿪱ٵĿռ��ָ��
		aW*aH * sizeof(float)    //����Ҫ���ٿռ���ֽ���
	);
	hipMalloc(
		(void**)&d_B,
		bW*bH * sizeof(float)
	);

	// �� �Դ� ��Ϊ��Ҫ����������ľ��󿪱ٿռ�
	hipMalloc(
		(void**)&d_C,
		cW*cH * sizeof(float)
	);

	// ���������ݴ��ݽ� �Դ� ���Ѿ����ٺ��˵Ŀռ�
	hipblasSetVector(
		aW*aH,    // Ҫ�����Դ��Ԫ�ظ���
		sizeof(float),    // ÿ��Ԫ�ش�С
		h_A,    // ��������ʼ��ַ
		1,    // ����Ԫ��֮��Ĵ洢���
		d_A,    // GPU ����ʼ��ַ
		1    // ����Ԫ��֮��Ĵ洢���
	);
	hipblasSetVector(
		bW*bH,
		sizeof(float),
		h_B,
		1,
		d_B,
		1
	);

	// ͬ������
	hipDeviceSynchronize();

	// ���ݽ�������˺����еĲ��������庬����ο������ֲᡣ
	float a = 1; float b = 0;
	// ������ˡ��ú�����Ȼ���������������������
	hipblasSgemm(
		handle,    // blas �����
		HIPBLAS_OP_N,    // ���� A d_B��ת��
		HIPBLAS_OP_N,    // ���� B  d_A��ת��
		bW,    // d_B, C ������
		aH,    // d_A, C ������
		bH,    // d_B �������� d_A ������
		&a,    // ����ʽ�� �� ֵ 1
		d_B,    // A ���Դ��еĵ�ַ
		bW,    // lda  ʹd_Bת��
		d_A,    // B ���Դ��еĵ�ַ
		aW,    // ldb ʹd_Aת��
		&b,    // ����ʽ�� �� ֵ
		d_C,    // C ���Դ��еĵ�ַ(�������)
		cW    // ldc
	);
	// ͬ������
	hipDeviceSynchronize();

	// �� �Դ� ��ȡ���������� �ڴ���ȥ
	hipblasGetVector(
		cW*cH,    //  Ҫȡ��Ԫ�صĸ���
		sizeof(float),    // ÿ��Ԫ�ش�С
		d_C,    // GPU ����ʼ��ַ
		1,    // ����Ԫ��֮��Ĵ洢���
		h_C,    // ��������ʼ��ַ
		1    // ����Ԫ��֮��Ĵ洢���
	);
	hipEventRecord(event_stop, 0);
	hipEventSynchronize(event_stop);
	hipEventElapsedTime(&elapsedTime, event_start, event_stop);
	// ͬ��device ��֤�������ȷ����
	hipDeviceSynchronize();
	printf("matrix size:%d * %d \n", sizeT,sizeT);
	printf("cuda event time = %lfms\n", elapsedTime);
	mul_cpu(h_A, h_B, cpu_C);

	// ��ӡ������
	//cout << "��������ת�� ( (A*B)��ת�� )��" << endl;

	//for (int i = 0; i < cW*cH; i++) {
	//	cout << h_C[i] << " ";
	//	if ((i + 1) % cW == 0) cout << endl;
	//}

	//
	//cout << "cpu��������" << endl;

	//for (int i = 0; i < cW*cH; i++) {
	//	cout << cpu_C[i] << " ";
	//	if ((i + 1) % cW == 0) cout << endl;
	//}
	float diff = check_diff(h_C, cpu_C);
	printf("diff is %.10f\n", diff);
	// �����ʹ�ù����ڴ�
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// �ͷ� CUBLAS �����
	hipblasDestroy(handle);
	return 0;
}

int main() {
	for (int i = 0; i < 10; i++) {
		int t = pow(2, i);
		sizeT = t;
		aW = sizeT;
		aH = sizeT;
		bW = sizeT;
		bH = sizeT;
		cW = bW;
		cH = aH;
		matrix();
		
	}
	return 0;
}